
#include <hip/hip_runtime.h>
#include<cstdio>
__global__ void set_dynamic_position(float *arr, float t)
{
	int threadID = threadIdx.x;
	int blockID = blockIdx.x;
	int threads_per_block = blockDim.x;
	int i = blockID * threads_per_block + threadID;

	if (threadID == 0 or threadID == 1 or threadID == 2)
	{
		arr[i] = arr[i] * t;
	}
}

void device_set_dynamic_position(hipStream_t stream, float *d_arr, float t)
{
	dim3 blocks_per_grid(4, 1, 1);
	dim3 threads_per_block(5, 1, 1);

	set_dynamic_position<<<blocks_per_grid, threads_per_block, 0, stream>>>(d_arr, t);
}
