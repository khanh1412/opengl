
#include <hip/hip_runtime.h>
#include<cstdio>
__global__ void set_dynamic_position(float t, float *arr)
{
	int threadID = threadIdx.x;
	int blockID = blockIdx.x;
	int threads_per_block = blockDim.x;
	int i = blockID * threads_per_block + threadID;

	if (threadID == 0 or threadID == 1 or threadID == 2)
	{
		arr[i] = arr[i] * t;
	}
}

void device_set_dynamic_position(float t, float *d_arr)
{
	std::printf("device.cc : t = %d\n", t);
	dim3 blocks_per_grid(4, 1, 1);
	dim3 threads_per_block(5, 1, 1);

	set_dynamic_position<<<blocks_per_grid, threads_per_block>>>(t, d_arr);
}
