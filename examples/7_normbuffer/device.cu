
#include <hip/hip_runtime.h>
#include<cstdio>
__global__ void set_dynamic_positions(float *arr, float t)
{
	int threadID = threadIdx.x;
	int blockID = blockIdx.x;
	int threads_per_block = blockDim.x;
	int i = blockID * threads_per_block + threadID;

	if (threadID == 0 or threadID == 1 or threadID == 2)
	{
		arr[i] = arr[i] * t;
	}
}
__global__ void set_dynamic_indices(unsigned int *d_all_ib, unsigned int *d_ib, int flip)
{
	if (flip == blockIdx.x)
	{
		d_ib[threadIdx.x] = d_all_ib[blockDim.x * blockIdx.x + threadIdx.x];
		d_ib[3 + threadIdx.x] = 0;
	}
}

void device_set_dynamic_positions(hipStream_t stream, float *d_arr, float t)
{
	dim3 blocks_per_grid(4, 1, 1);
	dim3 threads_per_block(5, 1, 1);

	set_dynamic_positions<<<blocks_per_grid, threads_per_block, 0, stream>>>(d_arr, t);
}
static int flip = 1;
void device_set_dynamic_indices(hipStream_t stream, unsigned int *d_all_ib, unsigned int *d_ib)
{
	flip = (flip + 1) % 2;
	dim3 blocks_per_grid(2, 1, 1);
	dim3 threads_per_block(3, 1, 1);

	set_dynamic_indices<<<blocks_per_grid, threads_per_block, 0, stream>>>(d_all_ib, d_ib, flip);


	unsigned int arr[6];
	hipStreamSynchronize(stream);
	hipMemcpyAsync(&(arr[0]), d_ib, 6*sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
	hipStreamSynchronize(stream);
	std::printf("\nflip = %d\n", flip);
	std::printf("%d %d %d\n", arr[0], arr[1], arr[2]);
	std::printf("%d %d %d\n", arr[3], arr[4], arr[5]);
}
